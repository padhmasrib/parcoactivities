#include <hip/hip_runtime.h>

#include <cmath>
#include <fstream>
#include <iostream>
#include <random>

double G = 6.674 * std::pow(10, -11);
// double G = 1;

struct simulation {
  size_t nbpart;

  std::vector<double> mass;

  // position
  std::vector<double> x;
  std::vector<double> y;
  std::vector<double> z;

  // velocity
  std::vector<double> vx;
  std::vector<double> vy;
  std::vector<double> vz;

  // force
  std::vector<double> fx;
  std::vector<double> fy;
  std::vector<double> fz;

  simulation(size_t nb)
      : nbpart(nb),
        mass(nb),
        x(nb),
        y(nb),
        z(nb),
        vx(nb),
        vy(nb),
        vz(nb),
        fx(nb),
        fy(nb),
        fz(nb) {}
};

void random_init(simulation& s) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution dismass(0.9, 1.);
  std::normal_distribution dispos(0., 1.);
  std::normal_distribution disvel(0., 1.);

  for (size_t i = 0; i < s.nbpart; ++i) {
    s.mass[i] = dismass(gen);

    s.x[i] = dispos(gen);
    s.y[i] = dispos(gen);
    s.z[i] = dispos(gen);
    s.z[i] = 0.;

    s.vx[i] = disvel(gen);
    s.vy[i] = disvel(gen);
    s.vz[i] = disvel(gen);
    s.vz[i] = 0.;
    s.vx[i] = s.y[i] * 1.5;
    s.vy[i] = -s.x[i] * 1.5;
  }

  return;
  // normalize velocity (using normalization found on some physicis blog)
  // double meanmass = 0;
  // double meanmassvx = 0;
  // double meanmassvy = 0;
  // double meanmassvz = 0;
  // for (size_t i = 0; i < s.nbpart; ++i) {
  //   meanmass += s.mass[i];
  //   meanmassvx += s.mass[i] * s.vx[i];
  //   meanmassvy += s.mass[i] * s.vy[i];
  //   meanmassvz += s.mass[i] * s.vz[i];
  // }
  // for (size_t i = 0; i < s.nbpart; ++i) {
  //   s.vx[i] -= meanmassvx / meanmass;
  //   s.vy[i] -= meanmassvy / meanmass;
  //   s.vz[i] -= meanmassvz / meanmass;
  // }
}

void init_solar(simulation& s) {
  enum Planets {
    SUN,
    MERCURY,
    VENUS,
    EARTH,
    MARS,
    JUPITER,
    SATURN,
    URANUS,
    NEPTUNE,
    MOON
  };
  s = simulation(10);

  // Masses in kg
  s.mass[SUN] = 1.9891 * std::pow(10, 30);
  s.mass[MERCURY] = 3.285 * std::pow(10, 23);
  s.mass[VENUS] = 4.867 * std::pow(10, 24);
  s.mass[EARTH] = 5.972 * std::pow(10, 24);
  s.mass[MARS] = 6.39 * std::pow(10, 23);
  s.mass[JUPITER] = 1.898 * std::pow(10, 27);
  s.mass[SATURN] = 5.683 * std::pow(10, 26);
  s.mass[URANUS] = 8.681 * std::pow(10, 25);
  s.mass[NEPTUNE] = 1.024 * std::pow(10, 26);
  s.mass[MOON] = 7.342 * std::pow(10, 22);

  // Positions (in meters) and velocities (in m/s)
  double AU = 1.496 * std::pow(10, 11);  // Astronomical Unit

  s.x = {0,          0.39 * AU,
         0.72 * AU,  1.0 * AU,
         1.52 * AU,  5.20 * AU,
         9.58 * AU,  19.22 * AU,
         30.05 * AU, 1.0 * AU + 3.844 * std::pow(10, 8)};
  s.y = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  s.z = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};

  s.vx = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
  s.vy = {0, 47870, 35020, 29780, 24130, 13070, 9680, 6800, 5430, 29780 + 1022};
  s.vz = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
}

// meant to update the force that from applies on to
void update_force(simulation& s, size_t from, size_t to) {
  double softening = .1;
  double dist_sq = std::pow(s.x[from] - s.x[to], 2) +
                   std::pow(s.y[from] - s.y[to], 2) +
                   std::pow(s.z[from] - s.z[to], 2);
  double F = G * s.mass[from] * s.mass[to] /
             (dist_sq + softening);  // that the strength of the force

  // direction
  double dx = s.x[from] - s.x[to];
  double dy = s.y[from] - s.y[to];
  double dz = s.z[from] - s.z[to];
  double norm = std::sqrt(dx * dx + dy * dy + dz * dz);

  dx = dx / norm;
  dy = dy / norm;
  dz = dz / norm;

  // apply force
  s.fx[to] += dx * F;
  s.fy[to] += dy * F;
  s.fz[to] += dz * F;
}


__global__ void reset_force(double* d_fx, double* d_fy, double* d_fz, int nbpart) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < nbpart) {  // reset force
    d_fx[i] = 0.0;
    d_fy[i] = 0.0;
    d_fz[i] = 0.0;
  }
}

// meant to update the force that from applies on to
__global__ void update_force(double* d_mass, double* d_x, double* d_y,
                             double* d_z, double* d_fx, double* d_fy,
                             double* d_fz, int nbpart) {
  int from = blockIdx.x * blockDim.x + threadIdx.x;
  double G = 6.674 * std::pow(10, -11);
  double softening = .1;

  for (size_t to = 0; to < nbpart; ++to) {
    if (from != to) {
      double dx = d_x[from] - d_x[to];
      double dy = d_y[from] - d_y[to];
      double dz = d_z[from] - d_z[to];
      double dist_sq = dx * dx + dy * dy + dz * dz;
      // that the strength of the force
      double F = G * d_mass[from] * d_mass[to] / (dist_sq + softening);

      double norm = std::sqrt(dist_sq);
      dx = dx / norm;
      dy = dy / norm;
      dz = dz / norm;

      // apply force
      d_fx[to] += dx * F;
      d_fy[to] += dy * F;
      d_fz[to] += dz * F;
    }
  }
}

// update particle velocities and positions
__global__ void update_velocities_positions(double* d_mass, double* d_x,
                                            double* d_y, double* d_z,
                                            double* d_vx, double* d_vy,
                                            double* d_vz, double* d_fx,
                                            double* d_fy, double* d_fz,
                                            int nbpart, double dt) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < nbpart) {
    d_vx[i] += d_fx[i] / d_mass[i] * dt;
    d_vy[i] += d_fy[i] / d_mass[i] * dt;
    d_vz[i] += d_fz[i] / d_mass[i] * dt;

    d_x[i] += d_vx[i] * dt;
    d_y[i] += d_vy[i] * dt;
    d_z[i] += d_vz[i] * dt;
  }
}

void dump_state(simulation& s) {
  std::cout << s.nbpart << '\t';
  for (size_t i = 0; i < s.nbpart; ++i) {
    std::cout << s.mass[i] << '\t';
    std::cout << s.x[i] << '\t' << s.y[i] << '\t' << s.z[i] << '\t';
    std::cout << s.vx[i] << '\t' << s.vy[i] << '\t' << s.vz[i] << '\t';
    std::cout << s.fx[i] << '\t' << s.fy[i] << '\t' << s.fz[i] << '\t';
  }
  std::cout << '\n';
}

void load_from_file(simulation& s, std::string filename) {
  std::ifstream in(filename);
  size_t nbpart;
  in >> nbpart;
  s = simulation(nbpart);
  for (size_t i = 0; i < s.nbpart; ++i) {
    in >> s.mass[i];
    in >> s.x[i] >> s.y[i] >> s.z[i];
    in >> s.vx[i] >> s.vy[i] >> s.vz[i];
    in >> s.fx[i] >> s.fy[i] >> s.fz[i];
  }
  if (!in.good()) throw "kaboom";
}

int main(int argc, char* argv[]) {
  if (argc != 6) {
    std::cerr << "usage: " << argv[0] << " <input> <dt> <nbstep> <printevery> <CUDAblockSz>"
              << "\n"
              << "input can be:" << "\n"
              << "a number (random initialization)" << "\n"
              << "planet (initialize with solar system)" << "\n"
              << "a filename (load from file in singleline tsv)" << "\n"
              << "CUDA block size" << "\n";
    return -1;
  }

  double dt = std::atof(argv[2]);  // in seconds
  size_t nbstep = std::atol(argv[3]);
  size_t printevery = std::atol(argv[4]);

  simulation s(1);

  { // parse command line
    size_t nbpart = std::atol(argv[1]);  // return 0 if not a number
    if (nbpart > 0) {
      s = simulation(nbpart);
      random_init(s);
    } else {
      std::string inputparam = argv[1];
      if (inputparam == "planet") {
        init_solar(s);
      } else {
        load_from_file(s, inputparam);
      }
    }
  }

  size_t arrsize = s.nbpart * sizeof(double);  // array allocation size needed
  size_t blocksz = std::atol(argv[5]);  // CUDA block size: num. threads per block
  size_t gridsz = (s.nbpart + blocksz - 1) / blocksz;  // num. blocks per grid

    // std::cout << "#%# Inp:  1-nbpart: " << s.nbpart << "  2-dt: " << dt << "  3-nbstep: ";
    // std::cout << nbstep << "  4-print: " << printevery << "  5-blocksz: " << blocksz << "  arrsize: ";
    // std::cout << arrsize << "  gridsz: " << gridsz << "  dbl-sz: " << sizeof(double) << "\n\n";

  double *d_mass, *d_x, *d_y, *d_z, *d_vx, *d_vy, *d_vz, *d_fx, *d_fy, *d_fz;

  // allocate memory for the vectors on the device
  hipMalloc((void**)&d_mass, arrsize);
  hipMalloc((void**)&d_x, arrsize);
  hipMalloc((void**)&d_y, arrsize);
  hipMalloc((void**)&d_z, arrsize);
  hipMalloc((void**)&d_vx, arrsize);
  hipMalloc((void**)&d_vy, arrsize);
  hipMalloc((void**)&d_vz, arrsize);
  hipMalloc((void**)&d_fx, arrsize);
  hipMalloc((void**)&d_fy, arrsize);
  hipMalloc((void**)&d_fz, arrsize);

    // std::cout << "#%# Finished Allocating device memory\n";

  // copy initial data to the GPU
  hipMemcpy(d_mass, s.mass.data(), arrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_x, s.x.data(), arrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_y, s.y.data(), arrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_z, s.z.data(), arrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_vx, s.vx.data(), arrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_vy, s.vy.data(), arrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_vz, s.vz.data(), arrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_fx, s.fx.data(), arrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_fy, s.fy.data(), arrsize, hipMemcpyHostToDevice);
  hipMemcpy(d_fz, s.fz.data(), arrsize, hipMemcpyHostToDevice);

    // std::cout << "#%# Finished cudaMemcpyHostToDevice\n";

  for (size_t step = 0; step < nbstep; step++) {
    if (step % printevery == 0) {
      // copy results back to the host periodically for output
      hipMemcpy(s.mass.data(), d_mass, arrsize, hipMemcpyDeviceToHost);
      hipMemcpy(s.x.data(), d_x, arrsize, hipMemcpyDeviceToHost);
      hipMemcpy(s.y.data(), d_y, arrsize, hipMemcpyDeviceToHost);
      hipMemcpy(s.z.data(), d_z, arrsize, hipMemcpyDeviceToHost);
      hipMemcpy(s.vx.data(), d_vx, arrsize, hipMemcpyDeviceToHost);
      hipMemcpy(s.vy.data(), d_vy, arrsize, hipMemcpyDeviceToHost);
      hipMemcpy(s.vz.data(), d_vz, arrsize, hipMemcpyDeviceToHost);
      hipMemcpy(s.fx.data(), d_fx, arrsize, hipMemcpyDeviceToHost);
      hipMemcpy(s.fy.data(), d_fy, arrsize, hipMemcpyDeviceToHost);
      hipMemcpy(s.fz.data(), d_fz, arrsize, hipMemcpyDeviceToHost);
        // std::cout << "#%# finished cudaMemcpyDeviceToHost\n";
      dump_state(s); // outputs the results
    }

    // reset and update force computaion in device
    reset_force<<<gridsz, blocksz>>>(d_fx, d_fy, d_fz, s.nbpart);
    hipDeviceSynchronize();
      // std::cout << "#%# finished reset_force\n";

    update_force<<<gridsz, blocksz>>>(d_mass, d_x, d_y, d_z, d_fx, d_fy, d_fz, s.nbpart);
    hipDeviceSynchronize();
      // std::cout << "#%# finished update_force\n";

    // update particle velocities and positions
    update_velocities_positions<<<gridsz, blocksz>>>(d_mass, d_x, d_y, d_z,
                         d_vx, d_vy, d_vz, d_fx, d_fy, d_fz, s.nbpart, dt);
    hipDeviceSynchronize();
      // std::cout << "#%# finished update_velocities_positions\n";
  }

  // dump_state(s);

  hipFree(d_x);  hipFree(d_y);  hipFree(d_z);
  hipFree(d_vx); hipFree(d_vy); hipFree(d_vz);
  hipFree(d_fx); hipFree(d_fy); hipFree(d_fz);
  hipFree(d_mass);
    // std::cout << "#%# finished cudaFree\n";

  return 0;
}
